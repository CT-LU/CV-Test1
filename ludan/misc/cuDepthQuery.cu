#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

texture<short, 2, hipReadModeElementType> imageTex;
short *d_image; // original image
short *d_phase;
short *d_confidence;
size_t src_pitch;
size_t phase_pitch;
size_t conf_pitch;

__global__
void d_splitImage(int phase_pitch, int conf_pitch,
	          short* d_confidence, short* d_phase)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;                                                                     
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	if(blockIdx.x & 1) {
		d_phase[y*phase_pitch + x - 8 - ((blockIdx.x>>1)<<3)] = tex2D(imageTex, x, y);
	} else {
		d_confidence[y*conf_pitch + x - ((blockIdx.x>>1)<<3)] = tex2D(imageTex, x, y);
	}	
#if 0 // cpu simulate gpu	
	for(int bidx = 0; bidx < matImageDual.cols/8; bidx++) {
		for(int bidy = 0; bidy < matImageDual.rows/16; bidy++) {

			for(int tidx = 0; tidx < 8; tidx++) {
				for(int tidy = 0; tidy < 16; tidy++) {

					int x = bidx*8 + tidx;
					int y = bidy*16 + tidy;

					if(bidx % 2) {
						phase.at<short>(y, x - 8 - (bidx/2)*8) = matImageDual.at<short>(y, x);
					} else {
						confidence.at<short>(y, x - (bidx/2)*8) = matImageDual.at<short>(y, x);
					}	
				}
			}

		}
	}	
#endif
}

int iDivUp(int a, int b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

extern "C"
void splitImage(int width, int height, short* confidence, short* phase)
{
	dim3 blockSize(8, 16);
	dim3 gridSize(iDivUp(width, blockSize.x), iDivUp(height, blockSize.y));
	
	d_splitImage<<<gridSize, blockSize>>>(phase_pitch/sizeof(short), conf_pitch/sizeof(short),
		       			      d_confidence, d_phase);

	// check if kernel execution generated an error
        getLastCudaError("Error: d_splitImage Kernel execution FAILED");

	// copy result back from global memory to array
	checkCudaErrors(hipMemcpy2D(confidence, sizeof(short)*width/2, d_confidence, conf_pitch,
                                     sizeof(short)*width/2, height, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy2D(phase, sizeof(short)*width/2, d_phase, phase_pitch,
                                     sizeof(short)*width/2, height, hipMemcpyDeviceToHost));
}	

extern "C"
void initTexture(int width, int height, short* h_image)
{
	checkCudaErrors(hipMemcpy2D(d_image, src_pitch, h_image, sizeof(short)*width,
				sizeof(short)*width, height, hipMemcpyHostToDevice));

	// Bind the array to the texture    
	hipChannelFormatDesc desc = hipCreateChannelDesc<short>();
	imageTex.addressMode[0] = hipAddressModeClamp;
	imageTex.addressMode[1] = hipAddressModeClamp;
	imageTex.filterMode     = hipFilterModePoint;
	imageTex.normalized     = false;

	checkCudaErrors(hipBindTexture2D(0, imageTex, d_image, desc, width, height, src_pitch)); 
}

extern "C"
void allocateMem(int width, int height)
{
	// copy image data to array
	checkCudaErrors(hipMallocPitch((void **)&d_phase, &phase_pitch, (width/2)*sizeof(short), height));
	checkCudaErrors(hipMallocPitch((void **)&d_confidence, &conf_pitch, (width/2)*sizeof(short), height));
	checkCudaErrors(hipMallocPitch((void **)&d_image, &src_pitch, sizeof(short)*width, height));
}

extern "C"
void freeDevice()
{
	checkCudaErrors(hipFree(d_image));
	checkCudaErrors(hipFree(d_phase));
	checkCudaErrors(hipFree(d_confidence));
}
