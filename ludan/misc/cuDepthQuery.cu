#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__constant__ float const_gaussian[64];   //gaussian array in device side

texture<short, 2, hipReadModeElementType> image_tex;
texture<float, 2, hipReadModeElementType> xmap_tex;
texture<float, 2, hipReadModeElementType> ymap_tex;
texture<float, 2, hipReadModeElementType> p1_tex;
texture<float, 2, hipReadModeElementType> p2_tex;

short *d_image; // original image
short *d_phase;
short *d_confidence;
short *d_tmp;
float *d_xmap;
float *d_ymap;
unsigned char* d_mask;
float *d_p1;
float *d_p2;

size_t src_pitch;
size_t phase_pitch;
size_t conf_pitch;
size_t xmap_pitch;
size_t ymap_pitch;
size_t tmp_pitch;
size_t mask_pitch;
size_t p1_pitch;
size_t p2_pitch;

#define iDivUp(a, b)(a % b != 0) ? (a / b + 1) : (a / b)
//-------------------------------------------------------------------------------

/*
 * Euclidean Distance (x, y, d) = exp((|x - y|/d)^2 / 2)
 */
__device__ float euclideanDistance(float4 a, float4 b, float d)
{

    float mod = (b.x - a.x) * (b.x - a.x) +
                (b.y - a.y) * (b.y - a.y) +
                (b.z - a.z) * (b.z - a.z);

    return __expf(-mod / (2.f * d * d));
}

/*
 *
 */
#if 0
__global__ void
d_bilateralFilter(short* dst, int dst_pitch,
                   float e_d,  int radius)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    float sum = 0.0f;
    float factor;
    float4 t = {0.f, 0.f, 0.f, 0.f};
    float4 center_pix = tex2D(rgbaTex, x, y);

    for (int i = -radius; i <= radius; i++)
    {
        for (int j = -radius; j <= radius; j++)
        {
            float4 current_pix = tex2D(rgbaTex, x + j, y + i);
            factor = const_gaussian[i + radius]*const_gaussian[j + radius]* 
                     euclideanDistance(current_pix, center_pix, e_d);            

            t += factor*current_pix;
            sum += factor;
        }
    }

    dst[y*dst_pitch + x] = rgbaFloatToInt(t/sum);
}
#endif
/*
 *
 */
__global__
void d_convert2Distance(unsigned char* mask, int mask_pitch,
	       	short* phase, int phase_pitch,
		float* P1, int P1_pitch,
		float* P2, int P2_pitch) {

	const int x = blockIdx.x*blockDim.x + threadIdx.x;
	const int y = blockIdx.y*blockDim.y + threadIdx.y;
					
	float pixel = phase[y*phase_pitch + x];

	pixel = (pixel - P2[y*P2_pitch + x])*P1[y*P1_pitch + x];

	short pixel_short = (short)(pixel*0.842 + 277.8459);
	
	if(mask[y*mask_pitch + x]) {
		pixel_short = 0;
	}		

	phase[y*phase_pitch + x] = pixel_short;
}

/*
 *
 */
__global__ 
void d_remap(int width, int height,
		short* dst, int dst_pitch,
		short* src, int src_pitch)
{
        const int x = blockIdx.x*blockDim.x + threadIdx.x;
        const int y = blockIdx.y*blockDim.y + threadIdx.y;
        
	float tmp_newx = tex2D(xmap_tex, x, y);							
	float tmp_newy = tex2D(ymap_tex, x, y);							

	int newx = (int)floor(tmp_newx + 0.5);
	int newy = (int)floor(tmp_newy + 0.5);

	if(newx < width && newy < height) {
		dst[y*dst_pitch + x] = src[newy*src_pitch + newx];
	} else {
		dst[y*dst_pitch + x] = src[y*src_pitch + x];
	}
}

/*
 *
 */
__global__
void d_splitImage(int phase_pitch, int conf_pitch,
	          short* d_confidence, short* d_phase)
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;                                                                     
	const int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	if(blockIdx.x & 1) {
		d_phase[y*phase_pitch + x - 8 - ((blockIdx.x>>1)<<3)] = tex2D(image_tex, x, y);
	} else {
		d_confidence[y*conf_pitch + x - ((blockIdx.x>>1)<<3)] = tex2D(image_tex, x, y);
	}	
#if 0 // cpu simulate gpu	
	for(int bidx = 0; bidx < matImageDual.cols/8; bidx++) {
		for(int bidy = 0; bidy < matImageDual.rows/16; bidy++) {

			for(int tidx = 0; tidx < 8; tidx++) {
				for(int tidy = 0; tidy < 16; tidy++) {

					int x = bidx*8 + tidx;
					int y = bidy*16 + tidy;

					if(bidx % 2) {
						phase.at<short>(y, x - 8 - (bidx/2)*8) = matImageDual.at<short>(y, x);
					} else {
						confidence.at<short>(y, x - (bidx/2)*8) = matImageDual.at<short>(y, x);
					}	
				}
			}

		}
	}	
#endif
}

/*
 *
 */
__global__
void d_makeMask(short* src, int src_pitch,
		unsigned char* mask, int mask_pitch) 
{
	const int x = blockIdx.x*blockDim.x + threadIdx.x;                                                                     
	const int y = blockIdx.y*blockDim.y + threadIdx.y;
	
	short pixel = src[y*src_pitch + x];

	if(pixel == 4095 || pixel <= 1)
		mask[y*mask_pitch + x] = 1;				
	else
		mask[y*mask_pitch + x] = 0;				
}

//--------------------------------------------------------------------------------------------------

/*
 *
 */
extern "C"
void convert2Distance(int width, int height, short* phase)
{
	dim3 blockSize(8, 16);
	dim3 gridSize(iDivUp(width, blockSize.x), iDivUp(height, blockSize.y));
	d_convert2Distance<<<gridSize, blockSize>>>(d_mask, mask_pitch/sizeof(unsigned char),
	       	 					d_tmp, tmp_pitch/sizeof(short),
							d_p1, p1_pitch/sizeof(float),
							d_p2, p2_pitch/sizeof(float)); 
	// copy result back from global memory to array
	checkCudaErrors(hipMemcpy2D(phase, sizeof(short)*width, d_tmp, tmp_pitch,
                                     sizeof(short)*width, height, hipMemcpyDeviceToHost));
}

/*
 *
 */
extern "C"
void splitImage(int width, int height, short* confidence, short* phase)
{
	dim3 blockSize(8, 16);
	dim3 gridSize(iDivUp(width, blockSize.x), iDivUp(height, blockSize.y));
	d_splitImage<<<gridSize, blockSize>>>(phase_pitch/sizeof(short), conf_pitch/sizeof(short),
		       			      d_confidence, d_phase);

	// copy result back from global memory to array
	checkCudaErrors(hipMemcpy2D(confidence, sizeof(short)*width/2, d_confidence, conf_pitch,
                                     sizeof(short)*width/2, height, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy2D(phase, sizeof(short)*width/2, d_phase, phase_pitch,
                                     sizeof(short)*width/2, height, hipMemcpyDeviceToHost));
}	

/*
 *
 */
extern "C"
void makeMask(int width, int height, unsigned char* dst)
{
	dim3 blockSize(8, 16);
	dim3 gridSize(iDivUp(width, blockSize.x), iDivUp(height, blockSize.y));
	d_makeMask<<<gridSize, blockSize>>>(d_tmp, tmp_pitch/sizeof(short),						
						d_mask, mask_pitch/sizeof(unsigned char));

	// copy result back from global memory to array
	checkCudaErrors(hipMemcpy2D(dst, sizeof(unsigned char)*width, d_mask, mask_pitch,
                                     sizeof(unsigned char)*width, height, hipMemcpyDeviceToHost));
}	

/*
 *
 */
extern "C"
void remapImage(int width, int height, short* dst)
{
	dim3 blockSize(8, 16);
	dim3 gridSize(iDivUp(width, blockSize.x), iDivUp(height, blockSize.y));
	d_remap<<<gridSize, blockSize>>>(width, height,
						d_tmp, tmp_pitch/sizeof(short),						
						d_phase, phase_pitch/sizeof(short));

	// copy result back from global memory to array
	checkCudaErrors(hipMemcpy2D(dst, sizeof(short)*width, d_tmp, tmp_pitch,
                                     sizeof(short)*width, height, hipMemcpyDeviceToHost));
}	

/*
*
*/
extern "C"
void initMapsTexture(int width, int height, float* h_xmap, float* h_ymap, float* h_p1, float* h_p2)
{
	checkCudaErrors(hipMemcpy2D(d_xmap, xmap_pitch, h_xmap, sizeof(float)*width,
				sizeof(float)*width, height, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy2D(d_ymap, ymap_pitch, h_ymap, sizeof(float)*width,
				sizeof(float)*width, height, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy2D(d_p1, p1_pitch, h_p1, sizeof(float)*width,
				sizeof(float)*width, height, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy2D(d_p2, p2_pitch, h_p2, sizeof(float)*width,
				sizeof(float)*width, height, hipMemcpyHostToDevice));

	// Bind the array to the texture    
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	xmap_tex.addressMode[0] = hipAddressModeClamp;
	xmap_tex.addressMode[1] = hipAddressModeClamp;
	xmap_tex.filterMode     = hipFilterModePoint;
	xmap_tex.normalized     = false;
	
	ymap_tex.addressMode[0] = hipAddressModeClamp;
	ymap_tex.addressMode[1] = hipAddressModeClamp;
	ymap_tex.filterMode     = hipFilterModePoint;
	ymap_tex.normalized     = false;
	
	p1_tex.addressMode[0] = hipAddressModeClamp;
	p1_tex.addressMode[1] = hipAddressModeClamp;
	p1_tex.filterMode     = hipFilterModePoint;
	p1_tex.normalized     = false;
	
	p2_tex.addressMode[0] = hipAddressModeClamp;
	p2_tex.addressMode[1] = hipAddressModeClamp;
	p2_tex.filterMode     = hipFilterModePoint;
	p2_tex.normalized     = false;
	
	checkCudaErrors(hipBindTexture2D(0, xmap_tex, d_xmap, desc, width, height, xmap_pitch)); 
	checkCudaErrors(hipBindTexture2D(0, ymap_tex, d_ymap, desc, width, height, ymap_pitch)); 
	checkCudaErrors(hipBindTexture2D(0, p1_tex, d_p1, desc, width, height, p1_pitch)); 
	checkCudaErrors(hipBindTexture2D(0, p2_tex, d_p2, desc, width, height, p2_pitch)); 
}

/*
 *
 */
extern "C"
void initImageTexture(int width, int height, short* h_image)
{
	checkCudaErrors(hipMemcpy2D(d_image, src_pitch, h_image, sizeof(short)*width,
				sizeof(short)*width, height, hipMemcpyHostToDevice));

	// Bind the array to the texture    
	hipChannelFormatDesc desc = hipCreateChannelDesc<short>();
	image_tex.addressMode[0] = hipAddressModeClamp;
	image_tex.addressMode[1] = hipAddressModeClamp;
	image_tex.filterMode     = hipFilterModePoint;
	image_tex.normalized     = false;

	checkCudaErrors(hipBindTexture2D(0, image_tex, d_image, desc, width, height, src_pitch)); 
}

/*
 *
 */
extern "C"
void updateGaussian(float delta, int radius)
{
    float gaussian[64];

    for (int i = 0; i < 2*radius + 1; ++i)
    {
        float x = i - radius;
        gaussian[i] = expf(-(x*x) / (2*delta*delta));
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(const_gaussian), gaussian, sizeof(float)*(2*radius+1)));
}

/*
 *
 */
extern "C"
void allocateMem(int width, int height)
{
	// copy image data to array
	checkCudaErrors(hipMallocPitch((void **)&d_phase, &phase_pitch, (width/2)*sizeof(short), height));
	checkCudaErrors(hipMallocPitch((void **)&d_confidence, &conf_pitch, (width/2)*sizeof(short), height));
	checkCudaErrors(hipMallocPitch((void **)&d_image, &src_pitch, width*sizeof(short), height));
	checkCudaErrors(hipMallocPitch((void **)&d_xmap, &xmap_pitch, (width/2)*sizeof(float), height));
	checkCudaErrors(hipMallocPitch((void **)&d_ymap, &ymap_pitch, (width/2)*sizeof(float), height));
	checkCudaErrors(hipMallocPitch((void **)&d_p1, &p1_pitch, (width/2)*sizeof(float), height));
	checkCudaErrors(hipMallocPitch((void **)&d_p2, &p2_pitch, (width/2)*sizeof(float), height));
	checkCudaErrors(hipMallocPitch((void **)&d_tmp, &tmp_pitch, (width/2)*sizeof(float), height));
	checkCudaErrors(hipMallocPitch((void **)&d_mask, &mask_pitch, (width/2)*sizeof(unsigned char), height));
}

/*
 *
 */
extern "C"
void freeDevice()
{
	checkCudaErrors(hipFree(d_image));
	checkCudaErrors(hipFree(d_phase));
	checkCudaErrors(hipFree(d_confidence));
	checkCudaErrors(hipFree(d_xmap));
	checkCudaErrors(hipFree(d_ymap));
	checkCudaErrors(hipFree(d_p1));
	checkCudaErrors(hipFree(d_p2));
	checkCudaErrors(hipFree(d_tmp));
	checkCudaErrors(hipFree(d_mask));
}

