#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization
#include <hip/hip_runtime.h>
#include "gmmWithCuda.h"


#define FRAME_SIZE		(FRAME_WIDTH*FRAME_HEIGHT*FRAME_CHANNELS)

//GMM parameter
#define ALPHA 0.00005
#define DEF_COVARIANCE  8.0
#define MAX_COVARIANCE  11.0
#define COVARIANCE_THRESHOLD (1.5*1.5)
#define DEF_WEIGHT 0.00005
#define MAX_GMM_COMPONENTS	3

typedef struct __align__(32)
{
	float3 pixel_mean[FRAME_WIDTH*FRAME_HEIGHT];
	float covariance[FRAME_WIDTH*FRAME_HEIGHT];
	float weight[FRAME_WIDTH*FRAME_HEIGHT];
} gaussian_model;

__device__ 
float3 operator+(const float3 &a, const float3 &b) {

	return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ 
float3 operator-(const float3 &a, const float3 &b) {

	return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ 
float3 operator*(const float3 &a, const float3 &b) {

	return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}

__device__ 
float3 operator*(const float a, const float3 &b) {

	return make_float3(a*b.x, a*b.y, a*b.z);
}

/*
 * frame is from camera, this function is to initialize the gaussian models
 * It only be invoked once at first
 */
__global__ void
initializeGmm(uchar3* frame, gaussian_model* components)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	components[0].pixel_mean[index] = make_float3(frame[index].x, frame[index].y, frame[index].z);
	components[0].covariance[index] = DEF_COVARIANCE;
	components[0].weight[index] = 1.0;

#pragma unroll
	for (int i = 1; i < MAX_GMM_COMPONENTS; i++) {
		components[i].weight[index] = 0.0;
	}
}

/*
 * frame is from camera, gmm_frame is output filtered by gmm, components always stay on gpu's global memory
 * each invoking performGmm will update the components, return a new gmm_frame
 */
__global__ void
performGmm(const uchar3* frame, unsigned char* gmm_frame, gaussian_model* components)
{
	const int index = blockDim.x * blockIdx.x + threadIdx.x;
	//GMM processing parameter
	bool isMatch = false;
	float sum_of_weight = 0.0;
	float sum_of_square_diff = 0.0;
	float covariance_runtime = 0.0;
	float3 pixel_value;
	float3 pixel_mean;
	float3 pixel_diff;	

	//reset node runtime point
	gaussian_model* current_component = NULL;

	//get BGR value from each pixel

	pixel_value = make_float3(frame[index].x, frame[index].y, frame[index].z);
	
	//Macthing current pixel for GMM
#pragma unroll
	for(int k = 0; k < MAX_GMM_COMPONENTS; k++) {
		
		current_component = &components[k]; // component from 0 to max
		
		if (current_component->weight[index] == 0) {
			continue;
		}

		if(!isMatch) {
			//Handle matching for each Gaussian Component
			pixel_mean = current_component->pixel_mean[index];
			
			//get diff
			pixel_diff = pixel_value - pixel_mean;
			
			//get covariance for current gaussian model
			covariance_runtime = current_component->covariance[index];

			//get sum of square diff for BGR
			float3 tmp = pixel_diff*pixel_diff;
			sum_of_square_diff = tmp.x + tmp.y + tmp.z;
			
			//judge match or unmatch for current gaussian component
			if( sum_of_square_diff <= (COVARIANCE_THRESHOLD*covariance_runtime*covariance_runtime) )
			{
				//Match current Gaussian component
				//Update weight
				current_component->weight[index] = (1-ALPHA)*(current_component->weight[index]) + ALPHA;
				//Update Gaussian Component
				//Update mean
				pixel_mean = pixel_mean + ALPHA*pixel_diff;
				current_component->pixel_mean[index] = pixel_mean;

				//get new diff
				pixel_diff = pixel_value - pixel_mean;
				
				//update new sum of square_diff	
				float3 tmp = pixel_diff*pixel_diff;
				sum_of_square_diff = tmp.x + tmp.y + tmp.z;

				//Update covariance let Rho = ALPHA
				if (covariance_runtime < MAX_COVARIANCE) {
					covariance_runtime = covariance_runtime + ALPHA*(sum_of_square_diff - covariance_runtime);
					current_component->covariance[index] = covariance_runtime;
				}

				//Set match flag
				isMatch = true;

			}		
		}

		if (!isMatch) {
			//UnMatch current Gaussian component
			current_component->weight[index] = (1-ALPHA)*(current_component->weight[index]);
		}

		//get sum of weight
		sum_of_weight += current_component->weight[index];

	} // the end of the k components

	//if there is no match in GMM, delete the least weight gaussian component
	if(!isMatch) {
		
		current_component = &components[0]; 
		int min_component = 0;
		float min_weight = current_component->weight[index];

#pragma unroll
		for (int gg = 1; gg < MAX_GMM_COMPONENTS; gg++) {
			current_component = &components[gg];
			if (current_component->weight[index] < min_weight) {
				min_weight = current_component->weight[index];
				min_component = gg;
			}
		}					
		
		current_component = &components[min_component];
		sum_of_weight -= current_component->weight[index];
		current_component->pixel_mean[index] = pixel_value;
		current_component->covariance[index] = DEF_COVARIANCE;
		current_component->weight[index] = DEF_WEIGHT;

		//record the new sum of weight
		sum_of_weight += current_component->weight[index];
	}
	//normalize the sum of weight to 1, if sum of weight < 0.9 or sum of weight > 1.2
	//do normalization
	if(sum_of_weight < 0.9 || sum_of_weight > 1.2)
	{
		//Normalize the weight for each Gaussian component
#pragma unroll
		for (int gg = 0; gg < MAX_GMM_COMPONENTS; gg++) {
			current_component = &components[gg];
			current_component->weight[index] /= sum_of_weight;
		}
	}
	//judge foreground or background for current pixel and set the result to the gmm_frame
	if(!isMatch){
		//UnMatch any one of background GMM
		gmm_frame[index] = 255;
	} else {
		gmm_frame[index] = 0;
	}
}

/*
 * for allocating device memory
 */
uchar3* d_frame = NULL;	
unsigned char* d_gmm_frame = NULL;
gaussian_model* d_components;

#define THREADS 256 

/*
 * cpu invoke gpu kernel to initialize gmm models 
 */
extern "C"
void gpu_initialize_gmm(const unsigned char* frame)
{
	/*
	 * for kernel grid and thread num 
	 */
	int threadsPerBlock = THREADS;
	int blocksPerGrid = (FRAME_WIDTH*FRAME_HEIGHT) / threadsPerBlock;

	hipError_t err = hipSuccess;
	
	err = hipMalloc((void **)&d_gmm_frame, FRAME_WIDTH*FRAME_HEIGHT);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device gmm frame (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_frame, FRAME_SIZE);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device frame (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_frame, frame, FRAME_SIZE, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy frame from host to device while initializing (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	if (hipMalloc((void **)&d_components, sizeof(gaussian_model)*MAX_GMM_COMPONENTS) != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate components (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	initializeGmm<<<blocksPerGrid, threadsPerBlock>>>(d_frame, d_components);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch initializeGmm kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

/*
 * cpu invoke gpu kernel to perform CUDA-GMM and get output frame filtered by CUDA-GMM
 */
extern "C"
void gpu_perform_gmm(const unsigned char* frame, unsigned char* gmm_frame)
{
	/*
	 * for kernel grid and thread num 
	 */
	int threadsPerBlock = THREADS;
	int blocksPerGrid = (FRAME_WIDTH*FRAME_HEIGHT) / threadsPerBlock;

	hipError_t err = hipSuccess;
	err = hipMemcpy(d_frame, frame, FRAME_SIZE, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy frame from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	performGmm<<<blocksPerGrid, threadsPerBlock>>>(d_frame, d_gmm_frame, d_components);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch performGmm kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(gmm_frame, d_gmm_frame, FRAME_WIDTH*FRAME_HEIGHT, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy gmm frame from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

}

extern "C"
void gpu_free_gmm()
{
	hipError_t err = hipFree(d_frame);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_gmm_frame);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	hipDeviceReset();
}


